#include "hip/hip_runtime.h"
#include "CudaBitmap2D.h"
#include <Utilities/CudaUtils.h>
#include <GraphicalResources/Bitmap2D.h>

#include "hip/hip_runtime.h"
#include ""

using namespace Redline;



__device__ __forceinline __forceinline__ inline uint2 computeThreadIndex2D() 
{
	uint2 result;
	result.x = threadIdx.x + blockIdx.x * blockDim.x;
	result.y = threadIdx.y + blockIdx.y * blockDim.y;
	return result;
}

__global__ void rotatekernel(CudaBitmapData data) 
{
	uint2 i = computeThreadIndex2D();

	if (i.x > data.Width || i.y > data.Height) 
	{
		return;
	}

	data[i] = data[i].RotateForSDLDisplay();
}

void Redline::CudaBitmap2D::RotateForDisplay()
{
	dim3 grid, block;
	ComputeDispatchSize(grid, block);

	rotatekernel << <grid, block >> > (Data);
	cudaChecked(hipGetLastError());
}

__global__ void clearKernel(CudaBitmapData data, Color clearColor)
{
	uint2 i = computeThreadIndex2D();
	if (i.x > data.Width || i.y > data.Height)
	{
		return;
	}

	data[i] = clearColor;
}


Redline::CudaBitmap2D::CudaBitmap2D(unsigned int width, unsigned int height)
{
	_dataSize = sizeof(char) * 4 * width * height;
	cudaChecked(hipMalloc(&DataPointer, _dataSize));

	Data = CudaBitmapData();
	Data.Data = (Color*)DataPointer;
	Data.Width = width;
	Data.Height = height;
}

Redline::CudaBitmap2D::~CudaBitmap2D()
{
	if (DataPointer != nullptr)
	{
		cudaChecked(hipFree(DataPointer));
	}
	DataPointer = nullptr;
}

void Redline::CudaBitmap2D::CopyToBitmap2D(std::shared_ptr<Bitmap2D> destination)
{
	void* copyDest = destination->Pixels;
	if (destination->Width != Data.Width) 
	{
		throw std::exception();
	}
	if (destination->Height != Data.Height)
	{
		throw std::exception();
	}


	//cudaChecked(hipDeviceSynchronize());
	cudaChecked(hipMemcpy(copyDest, DataPointer, _dataSize, hipMemcpyDeviceToHost));
}

void Redline::CudaBitmap2D::CudaClear(Color color)
{
	dim3 block;// (8, 8);

	//int gridx = (Data.Width / 8) + ((Data.Width % 8) > 0 ? 1 : 0);
	//i/nt gridy = (Data.Height / 8) + ((Data.Height % 8) > 0 ? 1 : 0);

	dim3 grid;// (gridx, gridy);

	ComputeDispatchSize(grid, block);

	clearKernel<<<grid, block >>>(Data, color);
	cudaChecked(hipGetLastError());

}

void Redline::CudaBitmap2D::ComputeDispatchSize(dim3& gridsOut, dim3& blocksOut)
{
	blocksOut = dim3(8, 8);

	int gridx = (Data.Width / 8) + ((Data.Width % 8) > 0 ? 1 : 0);
	int gridy = (Data.Height / 8) + ((Data.Height % 8) > 0 ? 1 : 0);

	gridsOut = dim3(gridx, gridy);
}

